
#include <hip/hip_runtime.h>

__global__ void cavity(int nx, int ny, int nt, int nit,
                       float dx, float dy, float dt, int rho, float nu,
                       int N,
                       float *u, float *v, float *p, float *b,
                       float *un, float *vn, float *pn) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i>=N) return;

    /*
    matrix
    0 1 ... nx-1
    nx nx+1 ... 2nx-1
    2nx 2nx+1 ... 3nx-1
    ...
    (ny-1)nx (ny-1)nx+1 ... nynx-1
    */

    bool top = (i < nx);
    bool bottom = (i > (ny-1)*nx-1);
    bool left = (i % nx == 0);
    bool right = ((i+1) % nx == 0);
    bool inside = not (top or bottom or left or right);

    if (inside) {
        b[i] = rho * (1 / dt *\
                     ((u[i+1] - u[i-1]) / (2 * dx) + (v[i+nx] - v[i-nx]) / (2 * dy)) -\
                     powf(((u[i+1] - u[i-1]) / (2 * dx)), 2.0) - 2 * ((u[i+nx] - u[i-nx]) / (2 * dy) *\
                     (v[i+1] - v[i-1]) / (2 * dx)) - powf(((v[i+nx] - v[i-nx]) / (2 * dy)), 2.0));
    }

    for (int it=0; it<nit; it++) {
        pn[i] = p[i];
        if (inside) {
            p[i] = (powf(dy, 2.0) * (pn[i+1] + pn[i-1]) +\
                    powf(dx, 2.0) * (pn[i+nx] + pn[i-nx]) -\
                    b[i] * powf(dx, 2.0) * powf(dy, 2.0))\
                    / (2 * (powf(dx, 2.0) + powf(dy, 2.0)));
        }
        if (right) p[i] = p[i-1];
        if (left) p[i] = p[i+1];
        if (top) p[i] = p[i+nx];
        if (bottom) p[i] = 0.0;
    }

    un[i] = u[i];
    vn[i] = v[i];

    if (inside) {
        u[i] = un[i] - un[i] * dt / dx * (un[i] - un[i - 1])\
                    - un[i] * dt / dy * (un[i] - un[i-nx])\
                    - dt / (2 * rho * dx) * (p[i+1] - p[i-1])\
                    + nu * dt / powf(dx, 2.0) * (un[i+1] - 2 * un[i] + un[i-1])\
                    + nu * dt / powf(dy, 2.0) * (un[i+nx] - 2 * un[i] + un[i-nx]);
        v[i] = vn[i] - vn[i] * dt / dx * (vn[i] - vn[i - 1])\
                    - vn[i] * dt / dy * (vn[i] - vn[i-nx])\
                    - dt / (2 * rho * dx) * (p[i+nx] - p[i-nx])\
                    + nu * dt / powf(dx, 2.0) * (vn[i+1] - 2 * vn[i] + vn[i-1])\
                    + nu * dt / powf(dy, 2.0) * (vn[i+nx] - 2 * vn[i] + vn[i-nx]);
    }

    if (left) u[i] = v[i] = 0.0;
    if (right) u[i] = v[i] = 0.0;
    if (top) u[i] = v[i] = 0.0;
    if (bottom) {u[i] = 1.0; v[i] = 0.0;}
}


int main() {
    const int nx = 41;
    const int ny = 41;
    const int nt = 500;
    const int nit = 50;
    float dx = 2 / (nx - 1);
    float dy = 2 / (ny - 1);
    const float dt = 0.01;
    const int rho = 1;
    const float nu = 0.02;

    const int N = nx * ny;
    const int M = 1024;

    float *u, *v, *p, *b;
    float *un, *vn, *pn;

    hipMallocManaged(&u, ny * nx * sizeof(float));
    hipMallocManaged(&v, ny * nx * sizeof(float));
    hipMallocManaged(&p, ny * nx * sizeof(float));
    hipMallocManaged(&b, ny * nx * sizeof(float));
    hipMallocManaged(&un, ny * nx * sizeof(float));
    hipMallocManaged(&vn, ny * nx * sizeof(float));
    hipMallocManaged(&pn, ny * nx * sizeof(float));

    for (int i=0; i<ny*nx; i++) {
        u[i] = 0.0;
        v[i] = 0.0;
        p[i] = 0.0;
        b[i] = 0.0;

    }

    for (int n=0; n<nt; n++) {
        cavity<<<(N+M-1)/M,M>>>(nx, ny, nt, nit,
                                dx, dy, dt, rho, nu,
                                N,
                                u, v, p, b,
                                un, vn, pn);
        hipDeviceSynchronize();
    }

    hipFree(u);
    hipFree(v);
    hipFree(p);
    hipFree(b);
    hipFree(un);
    hipFree(vn);
    hipFree(pn);

    return 0;
}